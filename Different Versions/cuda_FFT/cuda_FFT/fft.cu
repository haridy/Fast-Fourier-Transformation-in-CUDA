#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <windows.h>
#include <conio.h>

#define _USE_MATH_DEFINES
#include "math.h"


//#define N 16
//#define N 4194304
#define N 16777216



float data_real[N];
float data_imag[N];
int reverse[N];
__global__ void ppt2(float* data_real_d,float* data_imag_d)
{
	unsigned int tid=threadIdx.x+blockIdx.x*blockDim.x;
	unsigned int x=tid;
	unsigned int slash=8;
	float tmp;
	int offset=N/slash;

	unsigned int index=x;
	unsigned h = 0;
    int i;
     // loop through all the bits
    for(i = 0; i < __log2f(N); i++)
    {
          // add bit from value to 1 bit left shifted variable
        h = (h << 1) + (x & 1);
        // right shift bits by 1
        x >>= 1;
    }
	if(h>index)
	{tmp=data_real_d[index];
	data_real_d[index]=data_real_d[h];
	data_real_d[h]=tmp;

	tmp=data_imag_d[index];
	data_imag_d[index]=data_imag_d[h];
	data_imag_d[h]=tmp;}
	
	for(int j=1;j<slash;j++)
	{x=tid+offset*j;
	index=x;
	h = 0;
    
     // loop through all the bits
    for(i = 0; i < __log2f(N); i++)
    {
          // add bit from value to 1 bit left shifted variable
        h = (h << 1) + (x & 1);
        // right shift bits by 1
        x >>= 1;
    }

	if(h>index)
	{tmp=data_real_d[index];
	data_real_d[index]=data_real_d[h];
	data_real_d[h]=tmp;

	tmp=data_imag_d[index];
	data_imag_d[index]=data_imag_d[h];
	data_imag_d[h]=tmp;}
	}

}


__global__ void ppt(float* data_real_d,float* data_imag_d,int* reverse_d)
{
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	float tmp;
	int value;
	int slash=2;
	int offset=N/slash;
	for(int i=0;i<slash;i++)
	{	tid=tid+offset*i;
		value=reverse_d[tid];
		if(value>tid)
		{
		tmp=data_imag_d[tid];
		data_imag_d[tid]=data_imag_d[value];
		data_imag_d[value]=tmp;
	
		tmp=data_real_d[tid];
		data_real_d[tid]=data_real_d[value];
		data_real_d[value]=tmp;
		}
	}

}

__global__ void fft(float* data_real_d,float* data_imag_d,int p)
{	
	
	unsigned int x,block,sub,index;	
	float tw_real;
	float tw_imag;
	unsigned int power;
	float tmp;
	float real,real2,imag,imag2;
	index=threadIdx.x+blockIdx.x*blockDim.x;
		
		power=__powf(2,p);
		//determine which block the thread is in(not cuda block)
		//x=N/(power*2);
		x=N>>(p+1);
		//block=(index)/x;
		//tmp2=__log2f(x);
		block=index>>(int)(__log2f(x));
		//sub is the subscript of the array where the thread should get his element1 for processing
		//tmp2=__log2f(block);
		//tmp2=x<<tmp2;
		sub=index+x*block;
		//issue request for real parts
		 real=data_real_d[sub];
		 real2=data_real_d[sub+x];
		//compute twiddle factor
		//tmp=(index)%x;
		tmp=(index)&(x-1);
		tmp=(2*M_PI*tmp*power)/N;
		tw_real=cosf(tmp);
		tw_imag=-1*sinf(tmp);
		//issue request for imaginary parts
		imag=data_imag_d[sub];
		imag2=data_imag_d[sub+x];
		//butterfly real parts
		tmp=real+real2;
		real2=real-real2;
		real=tmp;
		//write back real results of butterfly,only this part is written because we still need to twiddle the other
		data_real_d[sub]=real;
		//butterfly imag part
		tmp=imag+imag2;
		imag2=imag-imag2;
		imag=tmp;
		//multiply by twiddle
		tmp=real2;
		real2=real2*tw_real-imag2*tw_imag;
		data_real_d[sub+x]=real2;
		imag2=tmp*tw_imag+imag2*tw_real;
		//write back imag result of butterfly
		data_imag_d[sub]=imag;
		data_imag_d[sub+x]=imag2;
}



void bit_reversal()
{
   long i,i1,j,k,i2;
   double c1,c2,tx,ty;
   i2 = N >> 1;
   j = 0;
   for (i=0;i<N-1;i++) {
      if (i < j) {
         tx = data_real[i];
         ty = data_imag[i];
         data_real[i] = data_real[j];
         data_imag[i] = data_imag[j];
         data_real[j] = tx;
         data_imag[j] = ty;
      }
      k = i2;
      while (k <= j) {
         j -= k;
         k >>= 1;
      }
      j += k;
   }
}


int main( int argc, char** argv) 
{
	for(int i=0;i<N;i++)
	{	
		if(i<N/2) 
		{data_real[i]=1;
		data_imag[i]=0;}
		else{
		data_real[i]=0;
		data_imag[i]=0;
		}
	}
	
unsigned int h,index;
	for(int j=0;j<N;j++)
	{
	
	index=j;
	h = 0;
    
     // loop through all the bits
    for(int i = 0; i < log2f(N); i++)
    {
          // add bit from value to 1 bit left shifted variable
        h = (h << 1) + (index & 1);
        // right shift bits by 1
        index >>= 1;
    }
	//store value of h
	reverse[j]=h;
	}
printf("reverse[0]=%d\n",reverse[01]);

	int passes=log((float)N)/log((float)2);
	float* data_real_d;
	float* data_imag_d;
	//int* reverse_d;
	hipMalloc((void**)&data_real_d,N*sizeof(float));
	hipMalloc((void**)&data_imag_d,N*sizeof(float));
	//hipMalloc((void**)&reverse_d,N*sizeof(int));
	hipMemcpy(data_real_d,data_real,sizeof(float)*N,hipMemcpyHostToDevice);
	hipMemcpy(data_imag_d,data_imag,sizeof(float)*N,hipMemcpyHostToDevice);
	//hipMemcpy(reverse_d,reverse,sizeof(int)*N,hipMemcpyHostToDevice);

	dim3 dimBlock(512,1,1);
	dim3 dimGrid(N/1024,1,1);
	hipDeviceSynchronize();
	long int before = GetTickCount();
//-----------------------	
	hipEvent_t start, stop; float time;
	hipEventCreate(&start);
	hipEventCreate(&stop); hipEventRecord( start, 0 );
	for(int i=0;i<passes;i++)
	{
		fft<<<dimGrid,dimBlock>>>(data_real_d,data_imag_d,i);
	}
	hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
printf("fft time=%f\n",time);
//------------------------
	hipEventCreate(&start);
	hipEventCreate(&stop); hipEventRecord( start, 0 );
//    ppt<<<dim3(N/1024,1,1),dimBlock>>>(data_real_d,data_imag_d,reverse_d);
	ppt2<<<dim3(N/4096,1),dimBlock>>>(data_real_d,data_imag_d);
	hipDeviceSynchronize();
	hipEventRecord( stop, 0 ); 
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
//-----------------
	long int after = GetTickCount();

	const char* err=hipGetErrorString(hipGetLastError());
	
	for(int i=0;i<40;i++)
	{printf("%c",err[i]);}
	printf("\n");
	printf("%d ms\n",after-before);


	hipMemcpy(data_real,data_real_d,4*N,hipMemcpyDeviceToHost);
	hipMemcpy(data_imag,data_imag_d,4*N,hipMemcpyDeviceToHost);
	hipFree(data_real_d);
	hipFree(data_imag_d);
	long int before2 = GetTickCount();
	//bit_reversal();	
	long int after2=GetTickCount();

for(int i=N-16;i<N;i++)
	{
		printf("data[%d]=%f + %f i\n",i,data_real[i],data_imag[i]);
	}


	printf("ppt time= %f ms\n",time);
	
		
//	_getch();



}

